#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// Includes
//-----------------------------------------------------------------------------
#pragma region

#include "cu-smallpt.hpp"

#pragma endregion

//-----------------------------------------------------------------------------
// Defines
//-----------------------------------------------------------------------------
#pragma region

#define REFRACTIVE_INDEX_OUT 1.0
#define REFRACTIVE_INDEX_IN  1.5

#pragma endregion

//-----------------------------------------------------------------------------
// Declarations and Definitions
//-----------------------------------------------------------------------------
namespace smallpt {

	//__constant__ Sphere dev_spheres[9];

	const Sphere g_spheres[] = {
		Sphere(1e5,  Vector3(1e5 + 1, 40.8, 81.6),   Vector3(),   Vector3(0.75,0.25,0.25), Reflection_t::Diffuse),	 //Left
		Sphere(1e5,  Vector3(-1e5 + 99, 40.8, 81.6), Vector3(),   Vector3(0.25,0.25,0.75), Reflection_t::Diffuse),	 //Right
		Sphere(1e5,  Vector3(50, 40.8, 1e5),         Vector3(),   Vector3(0.75),           Reflection_t::Diffuse),	 //Back
		Sphere(1e5,  Vector3(50, 40.8, -1e5 + 170),  Vector3(),   Vector3(),               Reflection_t::Diffuse),	 //Front
		Sphere(1e5,  Vector3(50, 1e5, 81.6),         Vector3(),   Vector3(0.75),           Reflection_t::Diffuse),	 //Bottom
		Sphere(1e5,  Vector3(50, -1e5 + 81.6, 81.6), Vector3(),   Vector3(0.75),           Reflection_t::Diffuse),	 //Top
		Sphere(16.5, Vector3(27, 16.5, 47),          Vector3(),   Vector3(0.999),          Reflection_t::Specular),	 //Mirror
		Sphere(16.5, Vector3(73, 16.5, 78),          Vector3(),   Vector3(0.999),          Reflection_t::Refractive),//Glass
		Sphere(600,	 Vector3(50, 681.6 - .27, 81.6), Vector3(12), Vector3(),               Reflection_t::Diffuse)	 //Light
	};

	__device__ inline bool Intersect(const Sphere *dev_spheres, size_t nb_spheres, 
		const Ray &ray, size_t &id) noexcept {
		
		bool hit = false;
		for (size_t i = 0; i < nb_spheres; ++i) {
			if (dev_spheres[i].Intersect(ray)) {
				hit = true;
				id = i;
			}
		}

		return hit;
	}

	__device__ inline bool Intersect(const Sphere *dev_spheres, size_t nb_spheres, 
		const Ray &ray) noexcept {
		
		for (size_t i = 0; i < nb_spheres; ++i) {
			if (dev_spheres[i].Intersect(ray)) {
				return true;
			}
		}

		return false;
	}

	__device__ static Vector3 Radiance(const Sphere *dev_spheres, size_t nb_spheres, 
		const Ray &ray, hiprandState *state) noexcept {
		
		Ray r = ray;
		Vector3 L;
		Vector3 F(1.0);

		while (true) {
			size_t id;
			if (!Intersect(dev_spheres, nb_spheres, r, id)) {
				return L;
			}

			const Sphere &shape = dev_spheres[id];
			const Vector3 p = r(r.m_tmax);
			const Vector3 n = Normalize(p - shape.m_p);

			L += F * shape.m_e;
			F *= shape.m_f;

			// Russian roulette
			if (r.m_depth > 4) {
				const double continue_probability = shape.m_f.Max();
				if (hiprand_uniform_double(state) >= continue_probability) {
					return L;
				}
				F /= continue_probability;
			}

			// Next path segment
			switch (shape.m_reflection_t) {
			
			case Reflection_t::Specular: {
				const Vector3 d = IdealSpecularReflect(r.m_d, n);
				r = Ray(p, d, EPSILON_SPHERE, INFINITY, r.m_depth + 1);
				break;
			}
			
			case Reflection_t::Refractive: {
				double pr;
				const Vector3 d = IdealSpecularTransmit(r.m_d, n, REFRACTIVE_INDEX_OUT, REFRACTIVE_INDEX_IN, pr, state);
				F *= pr;
				r = Ray(p, d, EPSILON_SPHERE, INFINITY, r.m_depth + 1);
				break;
			}
			
			default: {
				const Vector3 w = n.Dot(r.m_d) < 0 ? n : -n;
				const Vector3 u = Normalize((abs(w.m_x) > 0.1 ? Vector3(0.0, 1.0, 0.0) : Vector3(1.0, 0.0, 0.0)).Cross(w));
				const Vector3 v = w.Cross(u);

				const Vector3 sample_d = CosineWeightedSampleOnHemisphere(hiprand_uniform_double(state), hiprand_uniform_double(state));
				const Vector3 d = Normalize(sample_d.m_x * u + sample_d.m_y * v + sample_d.m_z * w);
				r = Ray(p, d, EPSILON_SPHERE, INFINITY, r.m_depth + 1);
			}
			}
		}
	}

	__global__ static void kernel(const Sphere *dev_spheres, size_t nb_spheres, 
		uint32_t w, uint32_t h, Vector3 *Ls, uint32_t nb_samples) {
		
		const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
		const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
		const uint32_t offset = x + y * blockDim.x * gridDim.x;

		if (x >= w || y >= h) {
			return;
		}

		// RNG
		hiprandState state;
		hiprand_init(offset, 0, 0, &state);

		const Vector3 eye  = Vector3(50, 52, 295.6);
		const Vector3 gaze = Normalize(Vector3(0, -0.042612, -1));
		const double fov = 0.5135;
		const Vector3 cx = Vector3(w * fov / h, 0.0, 0.0);
		const Vector3 cy = Normalize(cx.Cross(gaze)) * fov;

		for (size_t sy = 0, i = (h - 1 - y) * w + x; sy < 2; ++sy) { // 2 subpixel row

			for (size_t sx = 0; sx < 2; ++sx) { // 2 subpixel column

				Vector3 L;

				for (size_t s = 0; s < nb_samples; s++) { // samples per subpixel
					const double u1 = 2.0 * hiprand_uniform_double(&state);
					const double u2 = 2.0 * hiprand_uniform_double(&state);
					const double dx = u1 < 1 ? sqrt(u1) - 1.0 : 1.0 - sqrt(2.0 - u1);
					const double dy = u2 < 1 ? sqrt(u2) - 1.0 : 1.0 - sqrt(2.0 - u2);
					const Vector3 d = cx * (((sx + 0.5 + dx) / 2 + x) / w - 0.5) +
						              cy * (((sy + 0.5 + dy) / 2 + y) / h - 0.5) + gaze;
					
					L += Radiance(dev_spheres, nb_spheres, 
						Ray(eye + d * 130, Normalize(d), EPSILON_SPHERE), &state) 
						* (1.0 / nb_samples);
				}
				
				Ls[i] += 0.25 * Clamp(L);
			}
		}
	}

	static void Render(uint32_t nb_samples) noexcept {
		const uint32_t w = 1024;
		const uint32_t h = 768;
		const uint32_t nb_pixels = w * h;

		// Set up device memory
		//HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(dev_spheres), spheres, sizeof(spheres)) );
		Sphere *dev_spheres;
		HANDLE_ERROR(hipMalloc((void**)&dev_spheres, sizeof(g_spheres)));
		HANDLE_ERROR(hipMemcpy(dev_spheres, g_spheres, sizeof(g_spheres), hipMemcpyHostToDevice));
		Vector3 *dev_Ls;
		HANDLE_ERROR(hipMalloc((void**)&dev_Ls, nb_pixels * sizeof(Vector3)));
		HANDLE_ERROR(hipMemset(dev_Ls, 0, nb_pixels * sizeof(Vector3)));

		// Kernel execution
		const dim3 nblocks(w / 16, h / 16);
		const dim3 nthreads(16, 16);
		kernel<<<nblocks, nthreads>>>(dev_spheres, _countof(g_spheres), w, h, dev_Ls, nb_samples);

		// Set up host memory
		Vector3 *Ls = (Vector3 *)malloc(nb_pixels * sizeof(Vector3));
		// Transfer device -> host
		HANDLE_ERROR(hipMemcpy(Ls, dev_Ls, nb_pixels * sizeof(Vector3), hipMemcpyDeviceToHost));

		// Clean up device memory
		HANDLE_ERROR(hipFree(dev_Ls));
		HANDLE_ERROR(hipFree(dev_spheres));

		WritePPM(w, h, Ls);

		// Clean up host memory
		free(Ls);
	}
}

int main(int argc, char *argv[]) {
	const uint32_t nb_samples = (argc == 2) ? atoi(argv[1]) / 4 : 1;
	smallpt::Render(nb_samples);

	return 0;
}